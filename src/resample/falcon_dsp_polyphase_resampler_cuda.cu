#include "hip/hip_runtime.h"
/******************************************************************************
 *
 * MIT License
 *
 * Copyright (c) 2019 OrthogonalHawk
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to
 * deal in the Software without restriction, including without limitation the
 * rights to use, copy, modify, merge, publish, distribute, sublicense, and/or
 * sell copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
 * IN THE SOFTWARE.
 *
 *****************************************************************************/

/* The polyphase resampler implementation is based on code provided at
 *  https://github.com/staticfloat/upfirdn; which appears to be originally
 *  from https://sourceforge.net/motorola/upfirdn/home/Home/
 *
 * The Resampler interface is somewhat modified for the author's purposes, but
 *  the original copyright is provided here due to the heavy reuse from the
 *  original code.
 */

/*
Copyright (c) 2009, Motorola, Inc
All Rights Reserved.
Redistribution and use in source and binary forms, with or without 
modification, are permitted provided that the following conditions are
met:

* Redistributions of source code must retain the above copyright notice,
this list of conditions and the following disclaimer.

* Redistributions in binary form must reproduce the above copyright 
notice, this list of conditions and the following disclaimer in the 
documentation and/or other materials provided with the distribution.

* Neither the name of Motorola nor the names of its contributors may be 
used to endorse or promote products derived from this software without 
specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS 
IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO,  
THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR 
PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR 
CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, 
EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, 
PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR 
PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF 
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING 
NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS 
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

/******************************************************************************
 *
 * @file     falcon_dsp_polyphase_resampler_cuda.cu
 * @author   OrthogonalHawk
 * @date     22-Apr-2019
 *
 * @brief    CUDA implementation of a polyphase resampler.
 *
 * @section  DESCRIPTION
 *
 * Implements CUDA version of a polyphase resampler.
 *
 * @section  HISTORY
 *
 * 22-Apr-2019  OrthogonalHawk  File created.
 *
 *****************************************************************************/

/******************************************************************************
 *                               INCLUDE_FILES
 *****************************************************************************/

#include <iostream>
#include <hip/hip_complex.h>
#include <stdint.h>

#include "resample/falcon_dsp_polyphase_resampler_cuda.h"

/******************************************************************************
 *                                 CONSTANTS
 *****************************************************************************/

const uint32_t MAX_NUM_INPUT_SAMPLES_PER_CUDA_KERNEL = 1024;
const uint32_t MAX_NUM_CUDA_THREADS = 256;

/******************************************************************************
 *                              ENUMS & TYPEDEFS
 *****************************************************************************/

/******************************************************************************
 *                                  MACROS
 *****************************************************************************/

/******************************************************************************
 *                            CLASS IMPLEMENTATION
 *****************************************************************************/

namespace falcon_dsp
{
    template<class T, class C>
    falcon_dsp_polyphase_resampler_cuda<T, C>::falcon_dsp_polyphase_resampler_cuda(uint32_t up_rate, uint32_t down_rate, std::vector<C>& filter_coeffs)
      : falcon_dsp_polyphase_resampler<T, C>(up_rate, down_rate, filter_coeffs)
    {
        /* allocate CUDA unified memory space for filter coefficients */
        hipMallocManaged(&m_cuda_filter_coeffs, falcon_dsp_polyphase_resampler<T, C>::m_transposed_coeffs.size() * sizeof(C));
        
        /* allocate CUDA unified memory space for input and output data */
        m_max_num_cuda_input_samples = (MAX_NUM_INPUT_SAMPLES_PER_CUDA_KERNEL +
                                        falcon_dsp_polyphase_resampler<T, C>::m_coeffs_per_phase + 1);
        
        std::cout << "Maximum input samples: " << m_max_num_cuda_input_samples << std::endl;
        
        m_max_num_cuda_output_samples = falcon_dsp_polyphase_resampler<T, C>::needed_out_count(MAX_NUM_INPUT_SAMPLES_PER_CUDA_KERNEL);
        
        hipMallocManaged(&m_cuda_input_samples, m_max_num_cuda_input_samples * sizeof(T));
        hipMallocManaged(&m_cuda_output_samples, m_max_num_cuda_output_samples * sizeof(T));
    }
    
    template<class T, class C>
    falcon_dsp_polyphase_resampler_cuda<T, C>::~falcon_dsp_polyphase_resampler_cuda(void)
    {
        std::lock_guard<std::mutex> lock(falcon_dsp_polyphase_resampler<T, C>::m_mutex);
        
        if (m_cuda_filter_coeffs)
        {
            hipFree(m_cuda_filter_coeffs);
            m_cuda_filter_coeffs = nullptr;
        }
        
        if (m_cuda_input_samples)
        {
            hipFree(m_cuda_input_samples);
            m_cuda_input_samples = nullptr;
        }
        
        if (m_cuda_output_samples)
        {
            hipFree(m_cuda_output_samples);
            m_cuda_output_samples = nullptr;
        }
    }

    /* CUDA kernel function that resamples the input array */
    __global__
    void _polyphase_resampler_cuda(hipFloatComplex * in, uint32_t in_len, hipFloatComplex * out, uint32_t out_len,
                                   hipFloatComplex * coeffs, uint32_t coeffs_len, uint32_t coeffs_per_phase,
                                   int64_t start_x_idx, uint32_t start_t,
                                   uint32_t up_rate, uint32_t down_rate)
    {
        /* compute the thread index */
        uint32_t thread_index = blockIdx.x * blockDim.x + threadIdx.x;
        
        /* compute local thread variables */
        int64_t thread_x_idx = start_x_idx;
        uint32_t thread_t = start_t;
        
        for (uint32_t ii = 0; ii < thread_index; ++ii)
        {
            /* compute the next 'cycle' updates */
            thread_t += down_rate;
            thread_x_idx += thread_t / up_rate;
            thread_t %= up_rate;
        }
        
        /* apply the polyphase filter */
        hipFloatComplex acc;
        hipFloatComplex * coeff_ptr = coeffs + thread_t * coeffs_per_phase;
        int64_t x_back_idx = thread_x_idx - coeffs_per_phase + 1;
        while (x_back_idx <= thread_x_idx)
        {
            acc = hipCaddf(acc, hipCmulf(in[x_back_idx++], *(coeff_ptr++)));
        }
        
        /* set the output variable */
        out[thread_index] = acc;
    }
    
    /* specialized implementation */
    template<>
    int32_t falcon_dsp_polyphase_resampler_cuda<std::complex<float>, std::complex<float>>::apply(std::vector<input_type>& in, std::vector<output_type>& out)
    {
        std::lock_guard<std::mutex> lock(falcon_dsp_polyphase_resampler<std::complex<float>, std::complex<float>>::m_mutex);
        
        hipFloatComplex * cuda_filter_coeffs = static_cast<hipFloatComplex *>(m_cuda_filter_coeffs);
        hipFloatComplex * cuda_input_data = static_cast<hipFloatComplex *>(m_cuda_input_samples);
        hipFloatComplex * cuda_output_data = static_cast<hipFloatComplex *>(m_cuda_output_samples);
        
        out.clear();
        
        /* copy the filter coefficients into CUDA memory */
        for (uint32_t ii = 0;
             ii < falcon_dsp_polyphase_resampler<std::complex<float>, std::complex<float>>::m_transposed_coeffs.size();
             ++ii)
        {
            cuda_filter_coeffs[ii] =
                *(static_cast<hipFloatComplex *>(
                    static_cast<void *>(&m_transposed_coeffs[ii])));   
        }
        
        /* don't bother with running a CUDA/GPU implementation if the input size is not
         *  larger than the state vector */
        if (in.size() <= falcon_dsp_polyphase_resampler<std::complex<float>, std::complex<float>>::m_state.size())
        {
            return falcon_dsp_polyphase_resampler<std::complex<float>, std::complex<float>>::apply(in, out);  
        }
        
        /* x_idx points to the latest processed input sample */
        int64_t x_idx = falcon_dsp_polyphase_resampler<std::complex<float>, std::complex<float>>::m_xOffset;
        while (static_cast<uint64_t>(x_idx) < in.size())
        {
            bool required_state_array = false;
            
            /* setup variables for the cases where the next sample is handled in the CPU */
            output_type acc = output_type(0);
            typename std::vector<coeff_type>::iterator coeff_iter =
                falcon_dsp_polyphase_resampler<std::complex<float>, std::complex<float>>::m_transposed_coeffs.begin() +
                falcon_dsp_polyphase_resampler<std::complex<float>, std::complex<float>>::m_t * falcon_dsp_polyphase_resampler<std::complex<float>, std::complex<float>>::m_coeffs_per_phase;
            
            /* need to look back over the previous samples to compute the
             *  current filtered value */
            int64_t x_back_idx = x_idx - falcon_dsp_polyphase_resampler<std::complex<float>, std::complex<float>>::m_coeffs_per_phase + 1;
            int64_t offset = 0 - x_back_idx;
            
            /* values toward the beginning of the vector may require samples
             *  from the state array; handle these in C++/CPU */
            if (offset > 0)
            {
                required_state_array = true;
                
                /* need to draw from the state buffer */
                typename std::vector<std::complex<float>>::iterator state_iter = falcon_dsp_polyphase_resampler<std::complex<float>, std::complex<float>>::m_state.end() - offset;
                while (state_iter < falcon_dsp_polyphase_resampler<std::complex<float>, std::complex<float>>::m_state.end())
                {
                    acc += *(state_iter++) * *(coeff_iter++);
                }
                x_back_idx += offset;
            }
            
            /* if possible, handle multiple samples at once using CUDA. check for whether or not
             *  the state array was required as a way to detect samples at the beginning of the
             *  input vector */
            uint32_t num_samples, num_threads, new_t;
            if (!required_state_array &&
                falcon_dsp_polyphase_resampler_cuda<std::complex<float>, std::complex<float>>::compute_next_kernel_params(
                        x_idx, in.size(), num_samples, num_threads, new_t))
            {
                /* copy input samples into CUDA memory. note that this includes both the previous
                 *  samples required for filtering NUM_INPUT_SAMPLES_PER_CUDA_KERNEL (one for
                 *  each kernel thread) */
                for (uint32_t ii = 0;
                     ii < num_samples + falcon_dsp_polyphase_resampler<std::complex<float>, std::complex<float>>::m_coeffs_per_phase + 1;
                     ++ii)
                {
                    cuda_input_data[ii] = *(static_cast<hipFloatComplex *>(
                                              static_cast<void *>(&in[x_back_idx + ii])));
                }
                
                uint32_t num_thread_blocks = 1;
                _polyphase_resampler_cuda<<<num_thread_blocks, num_threads>>>(
                        cuda_input_data,
                        m_max_num_cuda_input_samples,
                        cuda_output_data,
                        m_max_num_cuda_output_samples,
                        cuda_filter_coeffs,
                        falcon_dsp_polyphase_resampler<std::complex<float>, std::complex<float>>::m_transposed_coeffs.size(),
                        falcon_dsp_polyphase_resampler<std::complex<float>, std::complex<float>>::m_coeffs_per_phase,
                        falcon_dsp_polyphase_resampler<std::complex<float>, std::complex<float>>::m_coeffs_per_phase + 1,
                        falcon_dsp_polyphase_resampler<std::complex<float>, std::complex<float>>::m_t,
                        falcon_dsp_polyphase_resampler<std::complex<float>, std::complex<float>>::m_up_rate,
                        falcon_dsp_polyphase_resampler<std::complex<float>, std::complex<float>>::m_down_rate);
                
                /* wait for GPU to finish before accessing on host */
                hipDeviceSynchronize();
                
                /* copy output samples out of CUDA memory */
                for (uint32_t ii = 0; ii < falcon_dsp_polyphase_resampler<std::complex<float>, std::complex<float>>::needed_out_count(num_samples); ++ii)
                {
                    out.push_back(*
                        static_cast<std::complex<float> *>(
                            static_cast<void *>(&cuda_output_data[ii])));
                }
                
                /* update tracking parameters */
                falcon_dsp_polyphase_resampler<std::complex<float>, std::complex<float>>::m_t = new_t;
                x_idx += num_samples;
            }
            else
            {
                /* either handling data at the beginning of the vector or close to the end;
                 *  just handle here in the CPU to keep things simpler */
                while (x_back_idx <= x_idx)
                {
                    acc += in[x_back_idx++] * *(coeff_iter++);
                }

                out.push_back(acc);
                falcon_dsp_polyphase_resampler<std::complex<float>, std::complex<float>>::m_t += falcon_dsp_polyphase_resampler<std::complex<float>, std::complex<float>>::m_down_rate;

                int64_t advance_amount = falcon_dsp_polyphase_resampler<std::complex<float>, std::complex<float>>::m_t / falcon_dsp_polyphase_resampler<std::complex<float>, std::complex<float>>::m_up_rate;
                x_idx += advance_amount;

                // which phase of the filter to use
                falcon_dsp_polyphase_resampler<std::complex<float>, std::complex<float>>::m_t %= falcon_dsp_polyphase_resampler<std::complex<float>, std::complex<float>>::m_up_rate;
            }
        }
        
        falcon_dsp_polyphase_resampler<std::complex<float>, std::complex<float>>::m_xOffset = x_idx - in.size();

        /* finished resampling; now update the state buffer so that future (assumed contiguous)
         *  input vectors can be resampled using these old samples.
         * 
         * based on the check at the start of this function, if the input size is less than
         *  the state buffer size the computation was passed over to the C++ implementation
         *  so there is no need to duplicate that handling here. */
        
        /* just copy last input samples into state buffer */
        for (uint64_t state_copy_idx = 0;
             state_copy_idx < falcon_dsp_polyphase_resampler<std::complex<float>, std::complex<float>>::m_state.size();
             ++state_copy_idx)
        {
            falcon_dsp_polyphase_resampler<std::complex<float>, std::complex<float>>::m_state[state_copy_idx] =
                in[in.size() - falcon_dsp_polyphase_resampler<std::complex<float>, std::complex<float>>::m_state.size() + state_copy_idx];   
        }
        
        /* number of samples computed */
        return out.size();
    }
                 
    template<class T, class C>
    bool falcon_dsp_polyphase_resampler_cuda<T, C>::compute_next_kernel_params(int64_t cur_x_idx, size_t in_size,
                                                                               uint32_t& num_in_samples, uint32_t& num_threads,
                                                                               uint32_t& new_t)
    {
        uint32_t local_t = falcon_dsp_polyphase_resampler<T, C>::m_t;
        int64_t local_x_idx = cur_x_idx;
        
        num_in_samples = 0;
        num_threads = 0;
        new_t = local_t;
        
        bool reached_limit = false;
        while (!reached_limit)
        {
            /* compute the next 'cycle' updates */
            local_t += falcon_dsp_polyphase_resampler<T, C>::m_down_rate;
            int64_t advance_amount = local_t / falcon_dsp_polyphase_resampler<T, C>::m_up_rate;
            local_x_idx += advance_amount;
            local_t %= falcon_dsp_polyphase_resampler<T, C>::m_up_rate;
            
            /* check for sufficient input space */
            if ((local_x_idx - cur_x_idx) < MAX_NUM_INPUT_SAMPLES_PER_CUDA_KERNEL)
            {
                num_in_samples = local_x_idx - cur_x_idx;
                num_threads++;
                new_t = local_t;
            }
            else
            {
                reached_limit = true;
            }
             
            /* check for the thread limit */
            if ((num_threads + 1) > MAX_NUM_CUDA_THREADS)
            {
                reached_limit = true;   
            }
        }
        
        return (num_in_samples > 0 && num_threads > 0);
    }
    
    /* force instantiation for specific types */
    template class falcon_dsp_polyphase_resampler_cuda<std::complex<float>, std::complex<float>>;
}

/******************************************************************************
 *                            CLASS IMPLEMENTATION
 *****************************************************************************/
