#include "hip/hip_runtime.h"
/******************************************************************************
 *
 * MIT License
 *
 * Copyright (c) 2019 OrthogonalHawk
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to
 * deal in the Software without restriction, including without limitation the
 * rights to use, copy, modify, merge, publish, distribute, sublicense, and/or
 * sell copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
 * IN THE SOFTWARE.
 *
 *****************************************************************************/

/* The polyphase resampler implementation is based on code provided at
 *  https://github.com/staticfloat/upfirdn; which appears to be originally
 *  from https://sourceforge.net/motorola/upfirdn/home/Home/
 *
 * The Resampler interface is somewhat modified for the author's purposes, but
 *  the original copyright is provided here due to the heavy reuse from the
 *  original code.
 */

/*
Copyright (c) 2009, Motorola, Inc
All Rights Reserved.
Redistribution and use in source and binary forms, with or without 
modification, are permitted provided that the following conditions are
met:

* Redistributions of source code must retain the above copyright notice,
this list of conditions and the following disclaimer.

* Redistributions in binary form must reproduce the above copyright 
notice, this list of conditions and the following disclaimer in the 
documentation and/or other materials provided with the distribution.

* Neither the name of Motorola nor the names of its contributors may be 
used to endorse or promote products derived from this software without 
specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS 
IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO,  
THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR 
PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR 
CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, 
EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, 
PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR 
PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF 
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING 
NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS 
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

/******************************************************************************
 *
 * @file     falcon_dsp_polyphase_resampler_cuda.cu
 * @author   OrthogonalHawk
 * @date     22-Apr-2019
 *
 * @brief    CUDA implementation of a polyphase resampler.
 *
 * @section  DESCRIPTION
 *
 * Implements CUDA version of a polyphase resampler.
 *
 * @section  HISTORY
 *
 * 22-Apr-2019  OrthogonalHawk  File created.
 *
 *****************************************************************************/

/******************************************************************************
 *                               INCLUDE_FILES
 *****************************************************************************/

#include <iostream>
#include <hip/hip_complex.h>
#include <numeric>
#include <stdint.h>

#include "resample/falcon_dsp_polyphase_resampler_cuda.h"
#include "utilities/falcon_dsp_utils.h"

/******************************************************************************
 *                                 CONSTANTS
 *****************************************************************************/

const uint32_t MAX_NUM_INPUT_SAMPLES_PER_CUDA_KERNEL = 16384 * 16;
const uint32_t MAX_NUM_SHARED_MEMORY_COEFFS_PER_THREAD_BLOCK = 256;

const uint32_t MAX_NUM_OUTPUTS_PER_CUDA_THREAD = 32;
const uint32_t MAX_NUM_CUDA_THREADS = 256;

/******************************************************************************
 *                              ENUMS & TYPEDEFS
 *****************************************************************************/

struct output_sample_s
{
    bool             active;
    hipFloatComplex   acc;
    hipFloatComplex * coeff_ptr;
    int64_t          data_start_idx;
    int64_t          data_stop_idx;
};
        
/******************************************************************************
 *                                  MACROS
 *****************************************************************************/

/******************************************************************************
 *                            CLASS IMPLEMENTATION
 *****************************************************************************/

namespace falcon_dsp
{
    template<class T, class C>
    falcon_dsp_polyphase_resampler_cuda<T, C>::falcon_dsp_polyphase_resampler_cuda(uint32_t up_rate, uint32_t down_rate, std::vector<C>& filter_coeffs)
      : falcon_dsp_polyphase_resampler<T, C>(up_rate, down_rate, filter_coeffs)
    {
        /* allocate CUDA unified memory space for filter coefficients */
        hipMallocManaged(&m_cuda_filter_coeffs, falcon_dsp_polyphase_resampler<T, C>::m_transposed_coeffs.size() * sizeof(C));
                  
        /* allocate CUDA unified memory space for input and output data */
        m_max_num_cuda_input_samples = (MAX_NUM_INPUT_SAMPLES_PER_CUDA_KERNEL +
                                        falcon_dsp_polyphase_resampler<T, C>::m_coeffs_per_phase + 1);
        
        m_max_num_cuda_output_samples = falcon_dsp_polyphase_resampler<T, C>::needed_out_count(MAX_NUM_INPUT_SAMPLES_PER_CUDA_KERNEL);
        
        hipMallocManaged(&m_cuda_input_samples, m_max_num_cuda_input_samples * sizeof(T));
        hipMallocManaged(&m_cuda_output_samples, m_max_num_cuda_output_samples * sizeof(T));

        /* calculate the average number of samples that are advanced for
         *  each output sample */
        m_avg_advance_per_output_sample = get_average_advance_in_samples();

        /* calculate the optimal number of outputs per CUDA thread. this value
         *  is chosen to minimize the number of repeated requests for the same
         *  input data from global memory and to prevent memory buffer collisions
         *  between CUDA threads in the same warp */
        m_num_outputs_per_cuda_thread = static_cast<uint32_t>(
                                            std::ceil(static_cast<float>(falcon_dsp::calculate_filter_delay_from_up_down_rates(filter_coeffs.size(), up_rate, down_rate)) /
                                                      static_cast<float>(falcon_dsp_polyphase_resampler_cuda<T, C>::m_avg_advance_per_output_sample)));

        if (m_num_outputs_per_cuda_thread > MAX_NUM_OUTPUTS_PER_CUDA_THREAD)
        {
            m_num_outputs_per_cuda_thread = MAX_NUM_OUTPUTS_PER_CUDA_THREAD;
        }
        
       /* change the shared memory size to 8 bytes per shared memory bank. this is so that we
        *  can better handle complex<float> data, which is natively 8 bytes in size */
       hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);
    }
    
    template<class T, class C>
    falcon_dsp_polyphase_resampler_cuda<T, C>::~falcon_dsp_polyphase_resampler_cuda(void)
    {
        std::lock_guard<std::mutex> lock(falcon_dsp_polyphase_resampler<T, C>::m_mutex);
        
        if (m_cuda_filter_coeffs)
        {
            hipFree(m_cuda_filter_coeffs);
            m_cuda_filter_coeffs = nullptr;
        }
        
        if (m_cuda_input_samples)
        {
            hipFree(m_cuda_input_samples);
            m_cuda_input_samples = nullptr;
        }
        
        if (m_cuda_output_samples)
        {
            hipFree(m_cuda_output_samples);
            m_cuda_output_samples = nullptr;
        }
    }

    /* CUDA kernel function that resamples the input array */
    __global__
    void _polyphase_resampler_cuda(hipFloatComplex * in, uint32_t in_len,
                                   hipFloatComplex * out, uint32_t out_len,
                                   hipFloatComplex * coeffs, uint32_t coeffs_len,
                                   uint32_t coeffs_per_phase,
                                   uint32_t num_outputs_per_cuda_thread,
                                   int64_t start_x_idx,
                                   uint32_t start_t,
                                   uint32_t start_output_idx,
                                   uint32_t up_rate,
                                   uint32_t down_rate)
    {
        __shared__ hipFloatComplex s_coeffs[MAX_NUM_SHARED_MEMORY_COEFFS_PER_THREAD_BLOCK];
        output_sample_s out_samples[MAX_NUM_OUTPUTS_PER_CUDA_THREAD];

        /* initialize the output sample data structures */
        for (uint32_t ii = 0;
             ii < num_outputs_per_cuda_thread && ii < MAX_NUM_OUTPUTS_PER_CUDA_THREAD;
             ++ii)
        {
            out_samples[ii].active = false;
            out_samples[ii].coeff_ptr = nullptr;
            out_samples[ii].data_start_idx = LONG_MAX;
            out_samples[ii].data_stop_idx = LONG_MAX;
            out_samples[ii].acc.x = 0;
            out_samples[ii].acc.y = 0;
        }

        /* compute the thread index */
        uint32_t thread_index = blockIdx.x * blockDim.x + threadIdx.x;

        /* copy coefficients to shared memory */
        if (threadIdx.x < coeffs_len)
        {
            s_coeffs[threadIdx.x] = coeffs[threadIdx.x];
        }
        __syncthreads();

        /* compute local thread variables */
        int64_t thread_x_idx = start_x_idx;
        uint32_t thread_t = start_t;
        
        int64_t thread_start_output_sample_idx = start_output_idx + (thread_index * num_outputs_per_cuda_thread);
        
        /* verify that this thread has at least one output to compute */
        if (thread_start_output_sample_idx >= out_len)
        {
            return;
        }
        
        for (int64_t out_sample_idx = start_output_idx;
             out_sample_idx < thread_start_output_sample_idx;
             ++out_sample_idx)
        {
            /* compute the next output sample 'cycle' updates */
            thread_t += down_rate;
            thread_x_idx += thread_t / up_rate;
            thread_t %= up_rate;
        }
        
        /* capture the FIRST output sample information */
        out_samples[0].active = true;
        out_samples[0].coeff_ptr = s_coeffs + thread_t * coeffs_per_phase;
        out_samples[0].data_start_idx = thread_x_idx - coeffs_per_phase + 1;
        out_samples[0].data_stop_idx = thread_x_idx;

        int64_t thread_data_start_idx = out_samples[0].data_start_idx;
        int64_t thread_data_stop_idx =  out_samples[0].data_stop_idx;
        
        /* ensure that the output sample is still within the range of the
         *  configured output data; there may be cases where some threads in
         *  a thread block are not needed. */
        for (uint32_t out_sample_idx = 1;
             out_sample_idx < num_outputs_per_cuda_thread && (out_sample_idx + thread_start_output_sample_idx) < out_len;
             ++out_sample_idx)
        {
            /* compute the next output sample 'cycle' updates */
            thread_t += down_rate;
            thread_x_idx += thread_t / up_rate;
            thread_t %= up_rate;
            
            out_samples[out_sample_idx].active = true;
            out_samples[out_sample_idx].coeff_ptr = s_coeffs + thread_t * coeffs_per_phase;
            out_samples[out_sample_idx].data_start_idx = thread_x_idx - coeffs_per_phase + 1;
            out_samples[out_sample_idx].data_stop_idx = thread_x_idx;
            
            thread_data_stop_idx = out_samples[out_sample_idx].data_stop_idx;
        }
        
        uint32_t first_active_out_sample = 0;
        hipFloatComplex next_x_val;
        for (int64_t x_idx = thread_data_start_idx; x_idx < thread_data_stop_idx; ++x_idx)
        {
            next_x_val = in[x_idx];
            for (uint32_t thread_out_sample_idx = first_active_out_sample;
                 thread_out_sample_idx < num_outputs_per_cuda_thread;
                 ++thread_out_sample_idx)
            {               
                /* we don't need to check whether x_idx is less than data_stop_idx
                 *  here because it's checked later and once x_idx is >= data_stop_idx
                 *  this output is 'disabled' and will no longer be assessed */
                if (x_idx >= out_samples[thread_out_sample_idx].data_start_idx)
                {                   
                    out_samples[thread_out_sample_idx].acc = hipCaddf(out_samples[thread_out_sample_idx].acc,
                                                                     hipCmulf(next_x_val, *(out_samples[thread_out_sample_idx].coeff_ptr++)));
                                                              
                    if ((out_samples[thread_out_sample_idx].data_stop_idx - 1) <= x_idx)
                    {
                        /* finished computing acc for this output */
                        first_active_out_sample++;
                    }
                }
                else
                {
                    break;
                }
            }
        }
        
        /* set the global outputs */
        uint64_t global_output_sample_idx_base = start_output_idx + (thread_index * num_outputs_per_cuda_thread);
        for (uint32_t out_sample_idx = 0;
             out_sample_idx < num_outputs_per_cuda_thread && out_samples[out_sample_idx].active;
             ++out_sample_idx)
        {
            out[global_output_sample_idx_base + out_sample_idx] = out_samples[out_sample_idx].acc;
        }
    }
    
    /* specialized implementation */
    template<>
    int32_t falcon_dsp_polyphase_resampler_cuda<std::complex<float>, std::complex<float>>::apply(std::vector<input_type>& in, std::vector<output_type>& out)
    {
        std::lock_guard<std::mutex> lock(falcon_dsp_polyphase_resampler<std::complex<float>, std::complex<float>>::m_mutex);
        
        /* copy the filter coefficients into CUDA memory */
        hipMemcpy(m_cuda_filter_coeffs,
                   m_transposed_coeffs.data(),
                   m_transposed_coeffs.size() * sizeof(std::complex<float>),
                   hipMemcpyHostToDevice );
        
        hipFloatComplex * cuda_filter_coeffs = static_cast<hipFloatComplex *>(m_cuda_filter_coeffs);
        hipFloatComplex * cuda_input_data = static_cast<hipFloatComplex *>(m_cuda_input_samples);
        hipFloatComplex * cuda_output_data = static_cast<hipFloatComplex *>(m_cuda_output_samples);
        
        /* clear out the output and allocate space for the resulting data */
        out.clear();
        out.reserve(needed_out_count(in.size()));
        for (uint32_t out_idx = 0;
             out_idx < needed_out_count(in.size());
             ++out_idx)
        {
            out.push_back(std::complex<float>(0, 0));
        }
        uint32_t cur_out_idx = 0;
        
        /* don't bother with running a CUDA/GPU implementation if the input size is not
         *  larger than the state vector */
        if (in.size() <= falcon_dsp_polyphase_resampler<std::complex<float>, std::complex<float>>::m_state.size())
        {
            return falcon_dsp_polyphase_resampler<std::complex<float>, std::complex<float>>::apply(in, out);  
        }
        
        /* copy all input samples into CUDA memory */
        for (uint32_t in_data_idx = 0; in_data_idx < in.size(); ++in_data_idx)
        {
            cuda_input_data[in_data_idx] = *(static_cast<hipFloatComplex *>(static_cast<void *>(&in[in_data_idx])));
        }
        
        /* x_idx points to the latest processed input sample */
        int64_t x_idx = falcon_dsp_polyphase_resampler<std::complex<float>, std::complex<float>>::m_xOffset;
        while (static_cast<uint64_t>(x_idx) < in.size())
        {
            bool required_state_array = false;
            
            /* setup variables for the cases where the next sample is handled in the CPU */
            output_type acc = output_type(0);
            typename std::vector<coeff_type>::iterator coeff_iter =
                m_transposed_coeffs.begin() + m_t * m_coeffs_per_phase;
            
            /* need to look back over the previous samples to compute the
             *  current filtered value */
            int64_t x_back_idx = x_idx - m_coeffs_per_phase + 1;
            int64_t offset = 0 - x_back_idx;
            
            /* values toward the beginning of the vector may require samples
             *  from the state array; handle these in C++/CPU */
            if (offset > 0)
            {
                required_state_array = true;
                
                /* need to draw from the state buffer */
                typename std::vector<std::complex<float>>::iterator state_iter = falcon_dsp_polyphase_resampler<std::complex<float>, std::complex<float>>::m_state.end() - offset;
                while (state_iter < falcon_dsp_polyphase_resampler<std::complex<float>, std::complex<float>>::m_state.end())
                {
                    acc += *(state_iter++) * *(coeff_iter++);
                }
                x_back_idx += offset;
            }
            
            /* if possible, handle multiple samples at once using CUDA. check for whether or not
             *  the state array was required as a way to detect samples at the beginning of the
             *  input vector */
            if (!required_state_array &&
                (out.size() - cur_out_idx) > 0 &&
                m_transposed_coeffs.size() < MAX_NUM_SHARED_MEMORY_COEFFS_PER_THREAD_BLOCK)
            {
                uint32_t num_thread_blocks = (out.size() - cur_out_idx) / (MAX_NUM_CUDA_THREADS * m_num_outputs_per_cuda_thread);
                if ((out.size() - cur_out_idx) % (MAX_NUM_CUDA_THREADS * m_num_outputs_per_cuda_thread) != 0)
                {
                    num_thread_blocks++;
                }
                
                printf("Num required thread blocks(%u threads):%u for %u output samples (%u per thread)\n",
                    MAX_NUM_CUDA_THREADS, num_thread_blocks,
                    static_cast<uint32_t>((out.size() - cur_out_idx)), m_num_outputs_per_cuda_thread);

                _polyphase_resampler_cuda<<<num_thread_blocks, MAX_NUM_CUDA_THREADS>>>(
                        cuda_input_data,
                        m_max_num_cuda_input_samples,
                        cuda_output_data,
                        m_max_num_cuda_output_samples,
                        cuda_filter_coeffs,
                        m_transposed_coeffs.size(),
                        m_coeffs_per_phase,
                        m_num_outputs_per_cuda_thread,
                        x_idx, /* x_start_idx */
                        m_t,
                        cur_out_idx,
                        m_up_rate,
                        m_down_rate);

                uint32_t num_outputs_from_thread_blocks, new_t;
                int64_t new_x_idx;
                compute_next_filter_params(x_idx, in.size(), m_t,
                                           (out.size() - cur_out_idx),
                                           num_outputs_from_thread_blocks,
                                           new_t, new_x_idx);

                /* wait for GPU to finish before accessing on host */
                hipDeviceSynchronize();
                
                /* copy output samples out of CUDA memory */
                hipMemcpy(out.data() + cur_out_idx,
                           cuda_output_data + cur_out_idx,
                           num_outputs_from_thread_blocks * sizeof(std::complex<float>),
                           hipMemcpyDeviceToHost);
                
                /* update tracking parameters */
                m_t = new_t;
                x_idx += new_x_idx;
                cur_out_idx += num_outputs_from_thread_blocks;
            }
            else
            {               
                /* either handling data at the beginning of the vector or close to the end;
                 *  just handle here in the CPU to keep things simpler */
                while (x_back_idx <= x_idx)
                {
                    acc += in[x_back_idx++] * *(coeff_iter++);
                }
                
                if (cur_out_idx == 20)
                {
                    printf("acc.re=%f acc.im=%f\n", acc.real(), acc.imag());
                }

                out[cur_out_idx++] = acc;
                falcon_dsp_polyphase_resampler<std::complex<float>, std::complex<float>>::m_t += falcon_dsp_polyphase_resampler<std::complex<float>, std::complex<float>>::m_down_rate;

                int64_t advance_amount = falcon_dsp_polyphase_resampler<std::complex<float>, std::complex<float>>::m_t / falcon_dsp_polyphase_resampler<std::complex<float>, std::complex<float>>::m_up_rate;
                x_idx += advance_amount;

                // which phase of the filter to use
                falcon_dsp_polyphase_resampler<std::complex<float>, std::complex<float>>::m_t %= falcon_dsp_polyphase_resampler<std::complex<float>, std::complex<float>>::m_up_rate;
            }
        }
        
        falcon_dsp_polyphase_resampler<std::complex<float>, std::complex<float>>::m_xOffset = x_idx - in.size();

        /* finished resampling; now update the state buffer so that future (assumed contiguous)
         *  input vectors can be resampled using these old samples.
         * 
         * based on the check at the start of this function, if the input size is less than
         *  the state buffer size the computation was passed over to the C++ implementation
         *  so there is no need to duplicate that handling here. */
        
        /* just copy last input samples into state buffer */
        for (uint64_t state_copy_idx = 0;
             state_copy_idx < falcon_dsp_polyphase_resampler<std::complex<float>, std::complex<float>>::m_state.size();
             ++state_copy_idx)
        {
            falcon_dsp_polyphase_resampler<std::complex<float>, std::complex<float>>::m_state[state_copy_idx] =
                in[in.size() - falcon_dsp_polyphase_resampler<std::complex<float>, std::complex<float>>::m_state.size() + state_copy_idx];   
        }
        
        /* number of samples computed */
        return out.size();
    }
    
    template<class T, class C>
    void falcon_dsp_polyphase_resampler_cuda<T, C>::compute_next_filter_params(int64_t cur_x_idx, size_t in_size, uint32_t cur_t,
                                                                               uint32_t max_out_samples,
                                                                               uint32_t& num_out_samples,
                                                                               uint32_t& new_t,
                                                                               int64_t& new_x_idx)
    {
        uint32_t local_t = cur_t;
        new_x_idx = cur_x_idx;

        num_out_samples = 0;
        new_t = local_t;
        
        while (num_out_samples < max_out_samples && new_x_idx < in_size)
        {
            /* compute the next 'cycle' updates */
            local_t += falcon_dsp_polyphase_resampler<T, C>::m_down_rate;
            int64_t advance_amount = local_t / falcon_dsp_polyphase_resampler<T, C>::m_up_rate;
            new_x_idx += advance_amount;
            local_t %= falcon_dsp_polyphase_resampler<T, C>::m_up_rate;
            
            /* increment trackers; assuming one output sample per thread */
            num_out_samples++;
            new_t = local_t;
        }
    }
    
    template<class T, class C>
    uint32_t falcon_dsp_polyphase_resampler_cuda<T, C>::get_average_advance_in_samples(void)
    {
        const uint32_t NUM_SAMPLES_TO_EVALUATE = 1e6;

        uint32_t local_t = falcon_dsp_polyphase_resampler<T, C>::m_t;
        int64_t  local_x_idx = 0;
        std::vector<uint32_t> local_advances;

        bool reached_limit = false;
        while (!reached_limit && local_x_idx < NUM_SAMPLES_TO_EVALUATE)
        {
            /* compute the next 'cycle' updates */
            local_t += falcon_dsp_polyphase_resampler<T, C>::m_down_rate;
            int64_t advance_amount = local_t / falcon_dsp_polyphase_resampler<T, C>::m_up_rate;
            local_t %= falcon_dsp_polyphase_resampler<T, C>::m_up_rate;
            local_x_idx += advance_amount;

            local_advances.push_back(advance_amount);
        }

        float accum_sum = static_cast<float>(std::accumulate(local_advances.begin(), local_advances.end(), 0));
        float advance_avg = accum_sum / static_cast<float>(local_advances.size());

        return static_cast<uint32_t>(std::ceil(advance_avg));
    }

    /* force instantiation for specific types */
    template class falcon_dsp_polyphase_resampler_cuda<std::complex<float>, std::complex<float>>;
}

/******************************************************************************
 *                            CLASS IMPLEMENTATION
 *****************************************************************************/
