#include "hip/hip_runtime.h"
/******************************************************************************
 *
 * MIT License
 *
 * Copyright (c) 2019 OrthogonalHawk
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to
 * deal in the Software without restriction, including without limitation the
 * rights to use, copy, modify, merge, publish, distribute, sublicense, and/or
 * sell copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
 * IN THE SOFTWARE.
 *
 *****************************************************************************/

/******************************************************************************
 *
 * @file     falcon_dsp_freq_shift_cuda.cu
 * @author   OrthogonalHawk
 * @date     04-Jun-2019
 *
 * @brief    Implements a CUDA-based time series frequency shift operation.
 *
 * @section  DESCRIPTION
 *
 * Implements the CUDA version of a time series frequency shift operation. Both
 *  a standalone function and a class-based tranform object are supported.
 *
 * @section  HISTORY
 *
 * 04-Jun-2019  OrthogonalHawk  File created.
 *
 *****************************************************************************/

/******************************************************************************
 *                               INCLUDE_FILES
 *****************************************************************************/

#include <iostream>
#include <stdint.h>

#include <hip/hip_complex.h>

#include "transform/falcon_dsp_transform.h"

/******************************************************************************
 *                                 CONSTANTS
 *****************************************************************************/

/******************************************************************************
 *                              ENUMS & TYPEDEFS
 *****************************************************************************/

/******************************************************************************
 *                                  MACROS
 *****************************************************************************/

/******************************************************************************
 *                            CLASS IMPLEMENTATION
 *****************************************************************************/

namespace falcon_dsp
{
    /* CUDA kernel function that applies a frequency shift */
    __global__
    void _freq_shift(uint32_t num_samples_handled_previously,
                     uint32_t time_shift_rollover_sample_idx,
                     double   angular_freq,
                     hipFloatComplex * data,
                     uint32_t data_size)
    {
        /* retrieve the data index that corresponds to this thread */
        uint32_t data_index = blockIdx.x * blockDim.x + threadIdx.x;
     
        /* catch the case where the input size is not an integer
         *  multiple of the thread block size */
        if (data_index > data_size)
        {
            return;
        }
        
        /* compute the time shift index for the current thread */
        uint64_t orig_time_shift_idx = num_samples_handled_previously + data_index;
        uint64_t time_shift_idx = orig_time_shift_idx;
        
        for (uint32_t ii = 0; ii < 1; ++ii)
        {
            time_shift_idx += ii;
            time_shift_idx %= time_shift_rollover_sample_idx;
            
            /* compute the frequency shift multiplier value */
            float angle = angular_freq * time_shift_idx;
            float real = cosf(angle);
            float imag = sinf(angle);
            
            /* create a CUDA complex variable to apply the freqency shift */
            hipFloatComplex shift;
            shift.x = real;
            shift.y = imag;

            /* apply the frequency shift in-place */
            data[data_index] = hipCmulf(data[data_index], shift);
        }
    }
    
    /* @brief CUDA implementation of a frequency shift vector operation.
     * @param[in] in_sample_rate_in_sps - input vector sample rate in samples
     *                                      per second.
     * @param[in] in                    - input vector
     * @param[in] freq_shift_in_hz      - amount to frequency shift in Hz
     * @param[out] out                  - frequency shifted vector
     * @return True if the input vector was frequency shifted as requested;
     *          false otherwise.
     */
    bool freq_shift_cuda(uint32_t in_sample_rate_in_sps, std::vector<std::complex<int16_t>>& in,
                         int32_t freq_shift_in_hz, std::vector<std::complex<int16_t>>& out)
    {
        falcon_dsp_freq_shift_cuda freq_shifter(in_sample_rate_in_sps, freq_shift_in_hz);
        return freq_shifter.apply(in, out);
    }
    
    falcon_dsp_freq_shift_cuda::falcon_dsp_freq_shift_cuda(uint32_t input_sample_rate_in_sps, int32_t freq_shift_in_hz)
      : falcon_dsp_freq_shift(input_sample_rate_in_sps, freq_shift_in_hz),
        m_cuda_data_vector(nullptr),
        m_max_num_cuda_input_samples(0)
    { }
    
    falcon_dsp_freq_shift_cuda::~falcon_dsp_freq_shift_cuda(void)
    {
        std::lock_guard<std::mutex> lock(std::mutex);
        
        /* clean up existing memory */
        if (m_cuda_data_vector != nullptr)
        {
            hipFree(m_cuda_data_vector);
            m_cuda_data_vector = nullptr;
        }
    }

    bool falcon_dsp_freq_shift_cuda::apply(std::vector<std::complex<int16_t>>& in, std::vector<std::complex<int16_t>>& out)
    {
        std::lock_guard<std::mutex> lock(std::mutex);
        
        /* clear the output data structure */
        out.clear();
        
        /* if there is enough space in previously allocated memory then use it; otherwise
         *  allocate a new memory buffer. it is left as a future feature to specify a maximum
         *  memory size and process the data in chunks instead of requiring enough GPU
         *  memory to process the whole vector at once */
        if (m_max_num_cuda_input_samples < in.size())
        {
            /* clean up existing memory */
            if (m_cuda_data_vector != nullptr)
            {
                hipFree(m_cuda_data_vector);
                m_cuda_data_vector = nullptr;
            }
            
            /* allocate CUDA unified memory space for the data to be transformed. note that space is
             *  reserved for std::complex<float> because this is what is supported in CUDA. it will
             *  be converted back to std::complex<int16_t> before it is returned to the user */
            hipMallocManaged(&m_cuda_data_vector, in.size() * sizeof(std::complex<float>));
            m_max_num_cuda_input_samples = in.size();
        }

        hipFloatComplex * cuda_data = static_cast<hipFloatComplex *>(m_cuda_data_vector);
        
        for (uint32_t ii = 0; ii < in.size(); ++ii)
        {
            /* copy input samples into CUDA memory */
            std::complex<float> val(in[ii].real(), in[ii].imag());
            cuda_data[ii] = *(static_cast<hipFloatComplex *>(static_cast<void *>(&val)));
        }
        
        /* run kernel on the GPU */
        uint32_t thread_block_size = 256;
        uint32_t num_thread_blocks = (in.size() + thread_block_size - 1) / thread_block_size;
        _freq_shift<<<num_thread_blocks, thread_block_size>>>(m_samples_handled,
                                                              m_calculated_rollover_sample_idx,
                                                              m_angular_freq,
                                                              cuda_data,
                                                              in.size());
        
        /* wait for GPU to finish before accessing on host */
        hipDeviceSynchronize();
        
        /* copy output samples out of CUDA memory */
        for (uint32_t ii = 0; ii < in.size(); ++ii)
        {
            void * void_ptr = static_cast<void *>(&cuda_data[ii]);
            if (void_ptr != nullptr)
            {   
                std::complex<float> * complex_float_ptr = static_cast<std::complex<float> *>(void_ptr);
                if (complex_float_ptr != nullptr)
                {
                    out.push_back(*complex_float_ptr);
                }
                else
                {
                    std::cout << "ERROR: Found nullptr complex_float_ptr for ii=" << ii << std::endl;
                }
            }
            else
            {
                std::cout << "ERROR: found nullptr reference to cuda_data[" << ii << "]" << std::endl;
            }
        }
        
        m_samples_handled += in.size();
        m_samples_handled = static_cast<uint32_t>(m_samples_handled) % m_calculated_rollover_sample_idx;
            
        return out.size() > 0;
    }
}
