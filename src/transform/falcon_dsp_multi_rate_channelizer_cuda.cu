/******************************************************************************
 *
 * MIT License
 *
 * Copyright (c) 2020 OrthogonalHawk
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to
 * deal in the Software without restriction, including without limitation the
 * rights to use, copy, modify, merge, publish, distribute, sublicense, and/or
 * sell copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
 * IN THE SOFTWARE.
 *
 *****************************************************************************/

/******************************************************************************
 *
 * @file     falcon_dsp_multi_rate_channelizer_cuda.cu
 * @author   OrthogonalHawk
 * @date     28-Jan-2020
 *
 * @brief    Signal processing transformation class and functions to implement
 *            a multi-rate channelizer in CUDA.
 *
 * @section  DESCRIPTION
 *
 * Implements a set of signal processing transformation functions and classes that
 *  together implement a multi-rate channelizer and filtering capability.
 *  Implementation uses CUDA to leverage GPU acceleration.
 *
 * @section  HISTORY
 *
 * 28-Jan-2020  OrthogonalHawk  File created.
 * 31-Jan-2020  OrthogonalHawk  Optionally use optimized resampler kernel for
 *                               a single output per thread.
 *
 *****************************************************************************/

/******************************************************************************
 *                               INCLUDE_FILES
 *****************************************************************************/

#include <iostream>
#include <memory>
#include <stdint.h>

#include "transform/falcon_dsp_multi_rate_channelizer_cuda.h"
#include "utilities/falcon_dsp_host_timer.h"

/******************************************************************************
 *                                 CONSTANTS
 *****************************************************************************/

const bool TIMING_LOGS_ENABLED = false;

const uint32_t MAX_NUM_INPUT_SAMPLES_FOR_MULTI_CHAN_FREQ_SHIFT_KERNEL = 4;
const uint32_t MAX_NUM_OUTPUT_SAMPLES_PER_THREAD_FOR_RESAMPLER_KERNEL = 1;
const uint32_t MAX_NUM_CUDA_THREADS = 1024;

/******************************************************************************
 *                              ENUMS & TYPEDEFS
 *****************************************************************************/

/******************************************************************************
 *                                  MACROS
 *****************************************************************************/

namespace falcon_dsp
{
    /******************************************************************************
     *                        FUNCTION IMPLEMENTATION
     *****************************************************************************/
    
    /******************************************************************************
     *                           CLASS IMPLEMENTATION
     *****************************************************************************/
    
    falcon_dsp_multi_rate_channelizer_cuda::internal_multi_rate_channelizer_channel_s::internal_multi_rate_channelizer_channel_s(const multi_rate_channelizer_channel_s& other)
      : d_freq_shifted_data(nullptr),
        freq_shifted_data_len(0),
        d_resample_coeffs(nullptr),
        resample_coeffs_len(0),
        d_resample_output_params(nullptr),
        resample_output_params_len(0),
        d_resampled_data(nullptr),
        resampled_data_len(0)
    {
        output_sample_rate_in_sps = other.output_sample_rate_in_sps;
        freq_shift_in_hz = other.freq_shift_in_hz;
        up_rate = other.up_rate;
        down_rate = other.down_rate;
        resample_filter_coeffs = other.resample_filter_coeffs;

        resampler_params.initialize(up_rate, down_rate, resample_filter_coeffs);
            
        /* the resampler coefficients are fixed, so allocate space for them here */
        if (resample_coeffs_len != resample_filter_coeffs.size())
        {
            if (d_resample_coeffs)
            {
                cudaErrChk(hipFree(d_resample_coeffs));
                d_resample_coeffs = nullptr;
                resample_coeffs_len = 0;
            }

            resample_coeffs_len = resample_filter_coeffs.size();
            cudaErrChkAssert(hipMallocManaged(&d_resample_coeffs,
                                               resample_coeffs_len * sizeof(std::complex<float>)));
            
            /* copy the coefficients to the GPU */
            cudaErrChkAssert(hipMemcpy(static_cast<void *>(d_resample_coeffs),
                                        static_cast<void *>(resample_filter_coeffs.data()),
                                        resample_filter_coeffs.size() * sizeof(std::complex<float>),
                                        hipMemcpyHostToDevice));
        }
    }
            
    falcon_dsp_multi_rate_channelizer_cuda::internal_multi_rate_channelizer_channel_s::~internal_multi_rate_channelizer_channel_s(void)
    {
        cleanup_memory();
    }
    
    uint32_t falcon_dsp_multi_rate_channelizer_cuda::internal_multi_rate_channelizer_channel_s::get_num_outputs_for_input(uint32_t input_vector_len)
    {
        /* compute how many outputs will be generated for input_vector_len inputs */
        uint64_t np = input_vector_len * static_cast<uint64_t>(resampler_params.up_rate);
        uint32_t need = np / resampler_params.down_rate;
        
        if ((resampler_params.coeff_phase + resampler_params.up_rate * resampler_params.xOffset) < (np % resampler_params.down_rate))
        {
            need++;
        }
        
        return need;
    }
    
    uint32_t falcon_dsp_multi_rate_channelizer_cuda::internal_multi_rate_channelizer_channel_s::get_num_resampler_thread_blocks(void)
    {
        return (resample_output_params_len / MAX_NUM_CUDA_THREADS);
    }
    
    uint32_t falcon_dsp_multi_rate_channelizer_cuda::internal_multi_rate_channelizer_channel_s::initialize(uint32_t input_vector_len)
    {
        /* allocate space for the frequency shifted version of the input data. note that this
         *  is also the input data for resampling so it has to take into account the 
         *  resampling state information. therefore, do NOT use the freq_shift_channel_s
         *  allocate_memory method since this will only pay attention to the input vector
         *  length and fail to account for the state information. instead, handle the memory
         *  allocation manually */
        if (freq_shifted_data_len != (input_vector_len + resampler_params.state.size()))
        {
            if (d_freq_shifted_data)
            {
                cudaErrChkAssert(hipFree(d_freq_shifted_data));
                d_freq_shifted_data = nullptr;
                freq_shifted_data_len = 0;
            }

            freq_shifted_data_len = input_vector_len + resampler_params.state.size();
            cudaErrChkAssert(hipMallocManaged(&d_freq_shifted_data,
                                               freq_shifted_data_len * sizeof(std::complex<float>)));
        }

        /* frequency shift output memory has been allocated so update the frequency shift
         *  output memory pointer. note that the pointer is set AFTER the state information */
        freq_shift_chan->out_data = d_freq_shifted_data + resampler_params.state.size();
        freq_shift_chan->out_data_len = freq_shifted_data_len - resampler_params.state.size();
        
        /* copy the resampler state vector into CUDA memory */
        cudaErrChkAssert(hipMemcpy(d_freq_shifted_data,
                                    resampler_params.state.data(),
                                    resampler_params.state.size() * sizeof(std::complex<float>),
                                    hipMemcpyHostToDevice));
        
        /* calculate the number of thread blocks that will be required for resampling */
        uint32_t expected_num_outputs = get_num_outputs_for_input(input_vector_len);
        int64_t resample_x_idx = resampler_params.xOffset;
        uint32_t num_outputs_per_resampler_thread_block =
                MAX_NUM_CUDA_THREADS * MAX_NUM_OUTPUT_SAMPLES_PER_THREAD_FOR_RESAMPLER_KERNEL;
        uint32_t num_resampler_thread_blocks = expected_num_outputs / num_outputs_per_resampler_thread_block;
        if (expected_num_outputs % num_outputs_per_resampler_thread_block != 0)
        {
            num_resampler_thread_blocks++;
        }
        
        /* pre-compute resample output parameters */
        uint32_t num_outputs_from_thread_blocks = 0;
        uint32_t new_coeff_phase = resampler_params.coeff_phase;
        int64_t new_x_idx = resample_x_idx;
        falcon_dsp::falcon_dsp_polyphase_resampler_cuda::compute_output_params(resampler_params.up_rate,
                                                                               resampler_params.down_rate,
                                                                               resampler_params.state.size(),
                                                                               input_vector_len,
                                                                               resampler_params.coeff_phase,
                                                                               expected_num_outputs,
                                                                               num_outputs_from_thread_blocks,
                                                                               new_coeff_phase,
                                                                               new_x_idx,
                                                                               resample_output_params);
        
        /* update the channel tracking information preemptively, assuming that if the user
         *  calls the initialize method the parameters will actually be used */
        resampler_params.coeff_phase = new_coeff_phase;
        resample_x_idx += new_x_idx;
        resampler_params.xOffset = resample_x_idx - input_vector_len;

        /* allocate space for the output parameters */
        if (resample_output_params_len != (MAX_NUM_CUDA_THREADS * MAX_NUM_OUTPUT_SAMPLES_PER_THREAD_FOR_RESAMPLER_KERNEL * num_resampler_thread_blocks))
        {
            if (d_resample_output_params)
            {
                cudaErrChkAssert(hipFree(d_resample_output_params));
                d_resample_output_params = nullptr;
                resample_output_params_len = 0;
            }
            
            resample_output_params_len = MAX_NUM_CUDA_THREADS * MAX_NUM_OUTPUT_SAMPLES_PER_THREAD_FOR_RESAMPLER_KERNEL * num_resampler_thread_blocks;
            cudaErrChkAssert(hipMallocManaged(&d_resample_output_params,
                                               resample_output_params_len *
                                                   sizeof(polyphase_resampler_output_params_s)));
        }
        
        /* copy the output parameters into CUDA memory; these are recomputed each time the kernel runs
         *  although it is hoped that the memory does not need to be reallocated each time... */
        cudaErrChkAssert(hipMemcpy(d_resample_output_params,
                                    resample_output_params.data(),
                                    resample_output_params.size() * sizeof(polyphase_resampler_output_params_s),
                                    hipMemcpyHostToDevice));
        
        /* allocate space for the resampled outputs */
        if (resampled_data_len != num_outputs_from_thread_blocks)
        {
            if (d_resampled_data)
            {
                cudaErrChkAssert(hipFree(d_resampled_data));
                d_resampled_data = nullptr;
                resampled_data_len = 0;
            }
            
            resampled_data_len = num_outputs_from_thread_blocks;
            cudaErrChkAssert(hipMallocManaged(&d_resampled_data,
                                               resampled_data_len * sizeof(std::complex<float>)));
        }
        
        return num_outputs_from_thread_blocks;
    }
            
    void falcon_dsp_multi_rate_channelizer_cuda::internal_multi_rate_channelizer_channel_s::cleanup_memory(void)
    {        
        if (d_resample_coeffs)
        {
            cudaErrChk(hipFree(d_resample_coeffs));
            d_resample_coeffs = nullptr;
            resample_coeffs_len = 0;
        }

        if (d_resample_output_params)
        {
            cudaErrChk(hipFree(d_resample_output_params));
            d_resample_output_params = nullptr;
            resample_output_params_len = 0;
        }

        if (d_resampled_data)
        {
            cudaErrChk(hipFree(d_resampled_data));
            d_resampled_data = nullptr;
            resampled_data_len = 0;
        }

        /* when the freq_shift_channel_s class destructs it will automatically free any
         *  memory that it still has access to. however, here we're 'lying' to the class
         *  about the true memory pointer because the frequency shift output is the
         *  resampler input and therefore must account for resampler state information.
         *  the frequency shift memory allocation and freeing is therefore the responsibility
         *  of the internal_multi_rate_channelizer_channel_s class */
        if (freq_shift_chan && freq_shift_chan->out_data)
        {
            /* no hipFree here; handled below */
            freq_shift_chan->out_data = nullptr;
            freq_shift_chan->out_data_len = 0;
        }
        
        /* cleanup the memory allocated for frequency shifted output / resampler input */
        if (d_freq_shifted_data)
        {
            cudaErrChk(hipFree(d_freq_shifted_data));
            d_freq_shifted_data = nullptr;
            freq_shifted_data_len = 0;
        }
    }
    
    falcon_dsp_multi_rate_channelizer_cuda::falcon_dsp_multi_rate_channelizer_cuda(void)
      : m_initialized(false),
        m_cuda_input_data(nullptr),
        m_max_num_input_samples(0),
        d_freq_shift_channels(nullptr)
    {
        /* change the shared memory size to 8 bytes per shared memory bank. this is so that we
         *  can better handle complex<float> data, which is natively 8 bytes in size */
        cudaErrChkAssert(hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte));
    }
    
    falcon_dsp_multi_rate_channelizer_cuda::~falcon_dsp_multi_rate_channelizer_cuda(void)
    {
        std::lock_guard<std::mutex> lock(std::mutex);
        
        m_channels.clear();
        
        /* cleanup CUDA memory that was reserved for frequency shift channel information */
        if (d_freq_shift_channels)
        {
            cudaErrChk(hipFree(d_freq_shift_channels));
            d_freq_shift_channels = nullptr;
        }
    }

    bool falcon_dsp_multi_rate_channelizer_cuda::initialize(uint32_t input_sample_rate,
                                                            std::vector<multi_rate_channelizer_channel_s> channels)
    {
        std::lock_guard<std::mutex> lock(std::mutex);

        /* sanity check the inputs and verify that the class has not already been initialized */
        if (input_sample_rate == 0 ||
            channels.size() == 0 ||
            m_initialized)
        {
            return false;
        }

        /* check each one of the requested channels is achievable */
        for (auto chan_iter : channels)
        {
            /* TODO */
        }

        /* initialize the requested channels */
        for (auto chan_iter : channels)
        {            
            std::unique_ptr<internal_multi_rate_channelizer_channel_s> new_chan =
                    std::make_unique<internal_multi_rate_channelizer_channel_s>(chan_iter);
   
            auto freq_shift_params = falcon_dsp_freq_shift::get_freq_shift_params(input_sample_rate,
                                                                                  chan_iter.freq_shift_in_hz);
            
            std::unique_ptr<freq_shift_channel_s> new_freq_shift_chan = std::make_unique<freq_shift_channel_s>();
            new_freq_shift_chan->time_shift_rollover_sample_idx = freq_shift_params.first;
            new_freq_shift_chan->angular_freq = freq_shift_params.second;

            new_chan->freq_shift_chan = std::move(new_freq_shift_chan);
            m_channels.push_back(std::move(new_chan));
        }

        /* allocate CUDA memory for the frequency shift channel information; the master copy
         *  is kept within the m_channels data structure, but it is copied to the device when
         *  the 'apply' method is invoked */
        cudaErrChkAssert(hipMallocManaged(&d_freq_shift_channels,
                                           m_channels.size() * sizeof(freq_shift_channel_s)));

        /* initialization complete */
        m_initialized = true;
        
        return m_initialized;
    }
    
    bool falcon_dsp_multi_rate_channelizer_cuda::apply(std::vector<std::complex<float>>& in,
                                                       std::vector<std::vector<std::complex<float>>>& out)
    {
        std::lock_guard<std::mutex> lock(std::mutex);
        
        /* clear the output data structures and resize so that they can hold
         *  the shifted and resampled data. */
        out.clear();
        out.resize(m_channels.size());
        
        /* allocate CUDA memory for the input samples */
        if (m_max_num_input_samples != in.size())
        {
            if (m_cuda_input_data)
            {
                cudaErrChkAssert(hipFree(m_cuda_input_data));
                m_cuda_input_data = nullptr;
                m_max_num_input_samples = 0;
            }

            cudaErrChkAssert(hipMallocManaged(&m_cuda_input_data,
                                               in.size() * sizeof(std::complex<float>)));
            m_max_num_input_samples = in.size();
        }
        
        /* allocate CUDA memory for the intermediate and output samples */
        for (uint32_t chan_idx = 0; chan_idx < m_channels.size(); ++chan_idx)
        {
            /* resize the outputs based on the actual number of outputs expected
             *  from the resampling kernel. note that by using resize() the vector
             *  size is now equal to the final output size without explicitly
             *  adding data to the vector, which means that we can add data
             *  directly into the vector data buffer without worrying about the 
             *  vector size getting mismatched with the buffer contents (provided
             *  that the promised number of samples are actually copied in...) */
            out[chan_idx].resize(m_channels[chan_idx]->initialize(in.size()));
        }

        /* copy the input data to the GPU */
        cudaErrChkAssert(hipMemcpy(static_cast<void *>(m_cuda_input_data),
                                    static_cast<void *>(in.data()),
                                    in.size() * sizeof(std::complex<float>),
                                    hipMemcpyHostToDevice));

        /* copy the frequency shift channel information to the GPU */
        for (uint32_t chan_idx = 0; chan_idx < m_channels.size(); ++chan_idx)
        {
            cudaErrChkAssert(hipMemcpy(static_cast<void *>(&d_freq_shift_channels[chan_idx]),
                                        static_cast<void *>(m_channels[chan_idx]->freq_shift_chan.get()),
                                        sizeof(freq_shift_channel_s),
                                        hipMemcpyHostToDevice));
        }
        
        /* calculate frequency shift kernel parameters */
        uint32_t num_samples_per_freq_shift_thread = MAX_NUM_INPUT_SAMPLES_FOR_MULTI_CHAN_FREQ_SHIFT_KERNEL;
        uint32_t samples_per_freq_shift_thread_block = num_samples_per_freq_shift_thread * MAX_NUM_CUDA_THREADS;
        uint32_t num_thread_blocks = (in.size() + samples_per_freq_shift_thread_block - 1) /
                                             samples_per_freq_shift_thread_block;

        uint32_t freq_shift_shared_memory_size_in_bytes = sizeof(freq_shift_channel_s) * m_channels.size();
        
        falcon_dsp::falcon_dsp_host_timer timer("FREQ_SHIFT KERNEL", TIMING_LOGS_ENABLED);

        /* run the frequency shift multi-channel kernel on the GPU */
        __freq_shift_multi_chan<<<num_thread_blocks, MAX_NUM_CUDA_THREADS, freq_shift_shared_memory_size_in_bytes>>>(
                            d_freq_shift_channels,
                            m_channels.size(),
                            num_samples_per_freq_shift_thread,
                            m_cuda_input_data,
                            m_max_num_input_samples);
            
        cudaErrChkAssert(hipPeekAtLastError());
            
        /* wait for GPU to finish frequency shifting */
        cudaErrChkAssert(hipDeviceSynchronize());

        timer.log_duration("FREQ_SHIFT Kernel Complete");

        /* frequency shifting complete; update the trackers */
        for (uint32_t chan_idx = 0; chan_idx < m_channels.size(); ++chan_idx)
        {
            m_channels[chan_idx]->freq_shift_chan->num_samples_handled += in.size();
            m_channels[chan_idx]->freq_shift_chan->num_samples_handled =
                static_cast<uint32_t>(m_channels[chan_idx]->freq_shift_chan->num_samples_handled) % m_channels[chan_idx]->freq_shift_chan->time_shift_rollover_sample_idx;
        }
            
        /* now resample each channel */
        for (uint32_t chan_idx = 0; chan_idx < m_channels.size(); ++chan_idx)
        {
            std::stringstream resample_timer_name;
            resample_timer_name << "RESAMP KERNEL " << chan_idx;
            falcon_dsp::falcon_dsp_host_timer resample_timer(resample_timer_name.str(), TIMING_LOGS_ENABLED);
        
            if (MAX_NUM_OUTPUT_SAMPLES_PER_THREAD_FOR_RESAMPLER_KERNEL == 1)
            {
                __polyphase_resampler_single_out<<<num_thread_blocks, MAX_NUM_CUDA_THREADS>>>(
                             m_channels[chan_idx]->d_resample_coeffs,
                             m_channels[chan_idx]->resample_coeffs_len,
                             m_channels[chan_idx]->d_resample_output_params,
                             m_channels[chan_idx]->resample_output_params_len,
                             m_channels[chan_idx]->d_freq_shifted_data,
                             m_channels[chan_idx]->freq_shifted_data_len,
                             m_channels[chan_idx]->d_resampled_data,
                             m_channels[chan_idx]->resampled_data_len,
                             m_channels[chan_idx]->resampler_params.coeffs_per_phase);
            }
            else
            {
                __polyphase_resampler_multi_out<<<num_thread_blocks, MAX_NUM_CUDA_THREADS>>>(
                             m_channels[chan_idx]->d_resample_coeffs,
                             m_channels[chan_idx]->resample_coeffs_len,
                             m_channels[chan_idx]->d_resample_output_params,
                             m_channels[chan_idx]->resample_output_params_len,
                             m_channels[chan_idx]->d_freq_shifted_data,
                             m_channels[chan_idx]->freq_shifted_data_len,
                             m_channels[chan_idx]->d_resampled_data,
                             m_channels[chan_idx]->resampled_data_len,
                             m_channels[chan_idx]->resampler_params.coeffs_per_phase,
                             MAX_NUM_OUTPUT_SAMPLES_PER_THREAD_FOR_RESAMPLER_KERNEL);
            }

            cudaErrChkAssert(hipPeekAtLastError());

            /* wait for GPU to finish before accessing on host */
            cudaErrChkAssert(hipDeviceSynchronize());
        
            resample_timer.log_duration("Resampling Complete");
            
            /* copy output samples out of CUDA memory */
            cudaErrChkAssert(hipMemcpy(out[chan_idx].data(),
                                        m_channels[chan_idx]->d_resampled_data,
                                        m_channels[chan_idx]->resampled_data_len * sizeof(std::complex<float>),
                                        hipMemcpyDeviceToHost));

            /* finished resampling; now update the resampler state buffer*/
            _manage_resampler_state(chan_idx, in.size());
        }

        return out.size() > 0;
    }

    void falcon_dsp_multi_rate_channelizer_cuda::reset_state(void)
    {
        for (uint32_t chan_idx = 0; chan_idx < m_channels.size(); ++chan_idx)
        {
            m_channels[chan_idx]->freq_shift_chan->reset_state();
            m_channels[chan_idx]->resampler_params.reset_state();
        }
    }

    void falcon_dsp_multi_rate_channelizer_cuda::_manage_resampler_state(uint32_t chan_idx, uint32_t input_vector_len)
    {
        /* find number of samples retained in buffer */
        int64_t retain = m_channels[chan_idx]->resampler_params.state.size() - input_vector_len;
        if (retain > 0)
        {
            /* for input_vector_len smaller than state buffer, copy end of buffer to beginning */
            copy(m_channels[chan_idx]->resampler_params.state.end() - retain,
                 m_channels[chan_idx]->resampler_params.state.end(),
                 m_channels[chan_idx]->resampler_params.state.begin());
            
            /* then, copy the entire (short) input to end of buffer */
            uint32_t in_idx = 0;
            for (uint64_t state_copy_idx = retain;
                 state_copy_idx < m_channels[chan_idx]->resampler_params.state.size();
                 ++state_copy_idx)
            {
                /* compute the next index to copy. note that here we need to account for the
                 *  state buffer padding that was added to the resampler input */
                uint32_t next_idx_to_copy = in_idx + m_channels[chan_idx]->resampler_params.state.size();
                
                /* copy over the state information */
                cudaErrChkAssert(hipMemcpy(m_channels[chan_idx]->resampler_params.state.data() + state_copy_idx,
                                            m_channels[chan_idx]->d_freq_shifted_data + next_idx_to_copy,
                                            sizeof(std::complex<float>),
                                            hipMemcpyDeviceToHost));
                
                /* keep working through the resampler input buffer */
                in_idx++;
            }
        }
        else
        {
            /* just copy last input samples into state buffer */
            for (uint64_t state_copy_idx = 0;
                 state_copy_idx < m_channels[chan_idx]->resampler_params.state.size();
                 ++state_copy_idx)
            {
                /* compute the next index to copy. note that here we need to account for the
                 *  state buffer padding that was added to the resampler input */
                uint32_t next_idx_to_copy = m_channels[chan_idx]->resampled_data_len -
                                                m_channels[chan_idx]->resampler_params.state.size() +
                                                state_copy_idx;
                
                cudaErrChkAssert(hipMemcpy(m_channels[chan_idx]->resampler_params.state.data() + state_copy_idx,
                                            m_channels[chan_idx]->d_resampled_data + next_idx_to_copy,
                                            sizeof(std::complex<float>),
                                            hipMemcpyDeviceToHost));
            }
        }
    }
}
